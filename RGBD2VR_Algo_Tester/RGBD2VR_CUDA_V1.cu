#include <hip/hip_runtime.h>


__global__ void generateStereoImage(const double* Depdata, const unsigned char* RGBdata, const unsigned char* MaskPtr,
    double* ImgTagLPtr, double* ImgTagRPtr, unsigned char* ImgOutPtr, double* MvImgLPtr, double* MvImgRPtr,
    int width, int height, double cx, double cy, double f, double k, double Baseline, double WL,
    int isRGBFill, int FWin)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double d = 10.0;
    for (int i = index; i < width * height; i += stride)
    {
        int x = i % width;
        int y = i / width;

        double DepPixel = Depdata[i];
        double D = DepPixel;
        int r = RGBdata[i * 3 + 0];
        int g = RGBdata[i * 3 + 1];
        int b = RGBdata[i * 3 + 2];

        if (MaskPtr[i] == 0 && D > 0)
        {
            double xI_l = d / D * (D * (x - cx) / f + Baseline * k);
            int I_l = static_cast<int>((1 + xI_l / WL) * width / 2);
            ImgTagLPtr[I_l + y * width] = I_l + y * width;
            ImgOutPtr[i * 3 + 0] = r;
            ImgOutPtr[i * 3 + 1] = g;
            ImgOutPtr[i * 3 + 2] = b;
            MvImgLPtr[i] = I_l;

            if (isRGBFill == 3)
            {
                if (FWin == 0)
                    continue;

                for (int fj = -FWin; fj <= FWin; fj++)
                {
                    for (int fi = -FWin; fi <= FWin; fi++)
                    {
                        int new_y = y + fj;
                        int new_x = x + fi;

                        if (new_y >= 0 && new_y < height && new_x >= 0 && new_x < width)
                        {
                            int maskIndex = new_y * width + new_x;
                            if (MaskPtr[maskIndex] == 0)
                            {
                                int imgOutIndex = (maskIndex * 3) + (fi * width + fj);
                                ImgOutPtr[imgOutIndex + 0] = r;
                                ImgOutPtr[imgOutIndex + 1] = g;
                                ImgOutPtr[imgOutIndex + 2] = b;
                            }
                        }
                    }
                }
            }
        }
    }
}

void runStereoImageGenerationCUDA(const double* Depdata, const unsigned char* RGBdata, const unsigned char* MaskPtr,
    double* ImgTagLPtr, double* ImgTagRPtr, unsigned char* ImgOutPtr, double* MvImgLPtr, double* MvImgRPtr,
    int width, int height, double cx, double cy, double f, double k, double Baseline, double WL,
    int isRGBFill, int FWin)
{
    // Allocate device memory
    double* dev_Depdata;
    unsigned char* dev_RGBdata;
    unsigned char* dev_MaskPtr;
    double* dev_ImgTagLPtr;
    double* dev_ImgTagRPtr;
    unsigned char* dev_ImgOutPtr;
    double* dev_MvImgLPtr;
    double* dev_MvImgRPtr;

    hipMalloc((void**)&dev_Depdata, width * height * sizeof(double));
    hipMalloc((void**)&dev_RGBdata, width * height * 3 * sizeof(unsigned char));
    hipMalloc((void**)&dev_MaskPtr, width * height * sizeof(unsigned char));
    hipMalloc((void**)&dev_ImgTagLPtr, width * height * sizeof(double));
    hipMalloc((void**)&dev_ImgTagRPtr, width * height * sizeof(double));
    hipMalloc((void**)&dev_ImgOutPtr, width * height * 3 * sizeof(unsigned char));
    hipMalloc((void**)&dev_MvImgLPtr, width * height * sizeof(double));
    hipMalloc((void**)&dev_MvImgRPtr, width * height * sizeof(double));

    // Copy data from host to device
    hipMemcpy(dev_Depdata, Depdata, width * height * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_RGBdata, RGBdata, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dev_MaskPtr, MaskPtr, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (width * height + threadsPerBlock - 1) / threadsPerBlock;
    //generateStereoImage<<<numBlocks, threadsPerBlock >>>(dev_Depdata, dev_RGBdata, dev_MaskPtr, dev_ImgTagLPtr, dev_ImgTagRPtr,
    //    dev_ImgOutPtr, dev_MvImgLPtr, dev_MvImgRPtr, width, height, cx, cy, f, k,
    //    Baseline, WL, isRGBFill, FWin);
    generateStereoImage<<<numBlocks, threadsPerBlock >>>(dev_Depdata, dev_RGBdata, dev_MaskPtr, dev_ImgTagLPtr, dev_ImgTagRPtr,
        dev_ImgOutPtr, dev_MvImgLPtr, dev_MvImgRPtr, width, height, cx, cy, f, k,
        Baseline, WL, isRGBFill, FWin);



    // Copy data from device to host
    hipMemcpy(ImgTagLPtr, dev_ImgTagLPtr, width * height * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ImgTagRPtr, dev_ImgTagRPtr, width * height * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ImgOutPtr, dev_ImgOutPtr, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(MvImgLPtr, dev_MvImgLPtr, width * height * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(MvImgRPtr, dev_MvImgRPtr, width * height * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_Depdata);
    hipFree(dev_RGBdata);
    hipFree(dev_MaskPtr);
    hipFree(dev_ImgTagLPtr);
    hipFree(dev_ImgTagRPtr);
    hipFree(dev_ImgOutPtr);
    hipFree(dev_MvImgLPtr);
    hipFree(dev_MvImgRPtr);
}
